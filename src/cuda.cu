#include "hip/hip_runtime.h"
#include "cuda.hpp"
#include "hipblas.h"
#include "hip/hip_runtime.h"

using namespace std;

int *r1, *r2, *r3, *c1, *c2, *c3;
float *denom, *var1, *var2;
hipblasHandle_t handle;

//Kernels
__global__
void kernel_matrix_add(float *m1, float *m2, float *m3, int *r1, int *c1, int *r2, int *c2, int *r3, int *c3) {
	int x = threadIdx.x;
	int y = threadIdx.y;
	m1[x * (*c1) + y] = m2[x * (*c2) + y] + m3[x * (*c3) + y];
}

__global__
void kernel_ReLU(float *m1, float *m2, int *r1, int *c1, int *r2, int *c2) {
	int x = threadIdx.x;
	int y = threadIdx.y;
	m1[x * (*c1) + y] = max( (float)0, m2[x * (*c2) + y] );
}

__global__
void kernel_softmax(float *m1, float *m2, int *r1, int *c1, int *r2, int *c2, float *denom) {
	int x = threadIdx.x;
	int y = threadIdx.y;
	m1[x * (*c1) + y] = exp(m2[x * (*c2) + y]) / *denom;
}

__global__
void kernel_cross_entropy_prime(float *da, float *a, float *o) {
	int x = threadIdx.x;
	da[x] = -((o[x] / a[x]) + ((1-o[x]) / (1 - a[x])));
}

__global__
void kernel_softmax_prime(float *da0, float *a1, float *da1, int *r1) {
	int x = threadIdx.x;
	da0[x] = 0;
	for (int i = 0; i < *r1; i++) {
		if (i == x) {
			da0[x] += da1[i] * (a1[i] - pow(a1[i], 2));
		} else {
			da0[x] += da1[i] * (-(a1[i] * a1[x]));
		}
	}
}

__global__
void kernel_ReLU_prime_biases(float *db, float *a, float *da) {
	int x = threadIdx.x;
	db[x] += da[x] * (a[x] != 0);
}

__global__
void kernel_ReLU_prime_others(float *da0, float *dw, float *w, float *a0, float *a1, float *da1, int *r1, int *r2) {
	int idx = (blockIdx.x * 1024 + threadIdx.x);
	int idxW;
	if (idx < *r1) {
		da0[idx] = 0;
		for (int i = 0; i < *r2; i++) {
			idxW = (i * (*r1)) + idx;
			if (a1[i] != 0) {
				da0[idx] += (da1[i] * w[idxW]);
				dw[idxW] += (da1[i] * a0[idx]);
			}
		}
	}
}

__global__
void kernel_gradient_descent_step(float *x, float *dx, float *var1, float *var2, int *r1) {
	int idx = (blockIdx.x * 1024 + threadIdx.x);
	if (idx < *r1) {
		x[idx] = x[idx] - ((*var1) * (dx[idx] / (*var2)));
		dx[idx] = 0;
	}
}

//Wrappers and... stuff


void cuda_matrix_mul(float **m1, float **m2, float **m3, int R2, int C2, int C3) {
	/**
	 * Alright so, here all the matrices are actually stored in row major. But cuBLAS uses matrices in column major. Now 
	 * I didn't want to go and change the entire code to store all the matrices in column major. SO, here I am just 
	 * passing the 2nd matrix as 1st and vice versa to the multiplication operation with inverted dimensions of the 
	 * matrices and voila, even though cuBLAS will store the result in column major, for us, our resultant matrix is 
	 * stored in row major form. 
	 * And I couldn't use HIPBLAS_OP_T's because that way it would've still stored the resultant in column major. 
	 * (Basically, I would've gotten the transpose of the actual result that I wanted).
	 */
	int lda=C3,ldb=C2,ldc=C3;
    const float alf = 1;
    const float bet = 0;
    const float *alpha = &alf;
    const float *beta = &bet;
    const float *A = *m3;
    const float *B = *m2;
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, C3, R2, C2, alpha, A, lda, B, ldb, beta, *m1, ldc);
    hipDeviceSynchronize();
}

void cuda_matrix_add(float **m1, float **m2, float **m3, int R1, int C1) {
	*r1 = R1;
	*r2 = R1;
	*r3 = R1;
	*c1 = C1;
	*c2 = C1;
	*c3 = C1;
	dim3 grid(1, 1, 1);
	dim3 block(*r1, *c1, 1);
	kernel_matrix_add <<<grid, block>>> (*m1, *m2, *m3, r1, c1, r2, c2, r3, c3);
	hipDeviceSynchronize();
}

void cuda_ReLU(float **m1, float **m2, int R1) {
	*r1 = R1;
	*r2 = R1;
	*c1 = 1;
	*c2 = 1;
	dim3 grid(1, 1, 1);
	dim3 block(*r1, *c1, 1);
	kernel_ReLU <<<grid, block>>> (*m1, *m2, r1, c1, r2, c2);
	hipDeviceSynchronize();
}

void cuda_softmax(float **m1, float **m2, int R1) {
	*r1 = R1;
	*r2 = R1;
	*c1 = 1;
	*c2 = 1;
	dim3 grid(1, 1, 1);
	dim3 block(*r1, *c1, 1);
	*denom = 0;
	for (int i = 0; i < *r1; i++) {
		*denom += exp((*m2)[i]);
	}
	kernel_softmax <<<grid, block>>> (*m1, *m2, r1, c1, r2, c2, denom);
	hipDeviceSynchronize();
}

void cuda_cross_entropy_prime(float **da, float **a, float **o, int R) {
	dim3 grid(1, 1, 1);
	dim3 block(R, 1, 1);
	kernel_cross_entropy_prime <<<grid, block>>> (*da, *a, *o);
	hipDeviceSynchronize();
}

void cuda_softmax_prime(float **da0, float **a1, float **da1, int R) {
	*r1 = R;
	dim3 grid(1, 1, 1);
	dim3 block(R, 1, 1);
	kernel_softmax_prime <<<grid, block>>> (*da0, *a1, *da1, r1);
	hipDeviceSynchronize();
}

void cuda_ReLU_prime_biases(float **db, float **a, float **da, int R) {
	dim3 grid(1, 1, 1);
	dim3 block(R, 1, 1);
	kernel_ReLU_prime_biases <<<grid, block>>> (*db, *a, *da);
	hipDeviceSynchronize();
}

void cuda_ReLU_prime_others(float **da0, float **dw, float **w, float **a0, float **a1, float **da1, int R1, int R2) {
	*r1 = R1;
	*r2 = R2;
	dim3 grid((((int)(R1 / 1024)) + 1), 1, 1);
	dim3 block(1024, 1, 1);
	kernel_ReLU_prime_others <<<grid, block>>> (*da0, *dw, *w, *a0, *a1, *da1, r1, r2);
	hipDeviceSynchronize();
}

void cuda_gradient_descent_step(float **x, float **dx, float lr, int bs, int R1) {
	*r1 = R1;
	*var1 = lr;
	*var2 = bs;
	dim3 grid((((int)(R1 / 1024)) + 1), 1, 1);
	dim3 block(1024, 1, 1);
	kernel_gradient_descent_step <<<grid, block>>> (*x, *dx, var1, var2, r1);
	hipDeviceSynchronize();
}

template <typename T>
void cuda_allocate(T **ptr, long long size) {
	hipMallocManaged(ptr, size * sizeof(T));
	hipMemset(*ptr, 0, size * sizeof(T));
}

template <typename T>
void cuda_delete(T **ptr) {
	hipFree(*ptr);
}

void cuda_copy_to_device(float **arr1, float **arr2, int size) {
	hipMemcpy(*arr1, *arr2, size * sizeof(float), hipMemcpyHostToDevice);
}

bool cuda_init() {
	cuda_allocate(&r1, 1);
	cuda_allocate(&r2, 1);
	cuda_allocate(&r3, 1);
	cuda_allocate(&c1, 1);
	cuda_allocate(&c2, 1);
	cuda_allocate(&c3, 1);
	cuda_allocate(&denom, 1);
	cuda_allocate(&var1, 1);
	cuda_allocate(&var2, 1);
	hipblasCreate(&handle);

	int deviceCount, device;
    int gpuDeviceCount = 0;
    struct hipDeviceProp_t properties;
    hipError_t cudaResultCode = hipGetDeviceCount(&deviceCount);
    if (cudaResultCode != hipSuccess) {
        deviceCount = 0;
    }
    for (device = 0; device < deviceCount; ++device) {
        hipGetDeviceProperties(&properties, device);
        if (properties.major != 9999) {
            ++gpuDeviceCount;
        }
    }
    return gpuDeviceCount;
}

void cuda_destroy() {
	cuda_delete(&r1);
	cuda_delete(&r2);
	cuda_delete(&r3);
	cuda_delete(&c1);
	cuda_delete(&c2);
	cuda_delete(&c3);
	cuda_delete(&denom);
	cuda_delete(&var1);
	cuda_delete(&var2);
	hipblasDestroy(handle);
}


template void cuda_allocate <char> (char**, long long);
template void cuda_allocate <int> (int**, long long);
template void cuda_allocate <float> (float**, long long);
template void cuda_allocate <double> (double**, long long);
template void cuda_allocate <long long> (long long**, long long);

template void cuda_delete <char> (char**);
template void cuda_delete <int> (int**);
template void cuda_delete <float> (float**);
template void cuda_delete <double> (double**);
template void cuda_delete <long long> (long long**);
